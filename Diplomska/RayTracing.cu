#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include "Parameters.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

struct RayPayload {
	float3 color;
	float3 emittedLight;
	float3 origin;
	float3 direction;
	float3 normal;
	float done;
	unsigned int seed;
};

extern "C" {
	__constant__ Params params;
}

static __forceinline__ __device__ void computeRay(uint3 idx, uint3 dim, float3& origin, float3& direction, float2& offset) {
	origin = params.camPosition;

	const mat4 inverseProjection = params.inverseProjection;
	const mat4 inverseView = params.inverseView;
	const float2 d = 2.0f * make_float2(
		(static_cast<float>(idx.x) + offset.x) / static_cast<float>(dim.x),
		(static_cast<float>(idx.y) + offset.y) / static_cast<float>(dim.y)
	) - 1.0f;

	float4 homogeniousDeviceCoords = make_float4(d.x, -d.y, 1.0f, 1.0f);
	float4 viewSpace = inverseProjection * homogeniousDeviceCoords;
	float4 worldSpace = inverseView * make_float4(viewSpace.x, viewSpace.y, viewSpace.z, 0.0f);

	direction = make_float3(normalize(worldSpace));
}

static __forceinline__ __device__ void storePayload(RayPayload payload) {
	optixSetPayload_0(floatAsUint(payload.color.x));
	optixSetPayload_1(floatAsUint(payload.color.y));
	optixSetPayload_2(floatAsUint(payload.color.z));

	optixSetPayload_3(floatAsUint(payload.emittedLight.x));
	optixSetPayload_4(floatAsUint(payload.emittedLight.y));
	optixSetPayload_5(floatAsUint(payload.emittedLight.z));

	optixSetPayload_6(floatAsUint(payload.origin.x));
	optixSetPayload_7(floatAsUint(payload.origin.y));
	optixSetPayload_8(floatAsUint(payload.origin.z));

	optixSetPayload_9(floatAsUint(payload.direction.x));
	optixSetPayload_10(floatAsUint(payload.direction.y));
	optixSetPayload_11(floatAsUint(payload.direction.z));

	optixSetPayload_12(floatAsUint(payload.normal.x));
	optixSetPayload_13(floatAsUint(payload.normal.y));
	optixSetPayload_14(floatAsUint(payload.normal.z));

	optixSetPayload_15(floatAsUint(payload.done));
	optixSetPayload_16(payload.seed);
}

static __forceinline__ __device__ RayPayload loadPayload() {
	RayPayload payload = {};

	payload.color = make_float3(uintAsFloat(optixGetPayload_0()), uintAsFloat(optixGetPayload_1()), uintAsFloat(optixGetPayload_2()));
	payload.emittedLight = make_float3(uintAsFloat(optixGetPayload_3()), uintAsFloat(optixGetPayload_4()), uintAsFloat(optixGetPayload_5()));
	payload.origin = make_float3(uintAsFloat(optixGetPayload_6()), uintAsFloat(optixGetPayload_7()), uintAsFloat(optixGetPayload_8()));
	payload.direction = make_float3(uintAsFloat(optixGetPayload_9()), uintAsFloat(optixGetPayload_10()), uintAsFloat(optixGetPayload_11()));
	payload.normal = make_float3(uintAsFloat(optixGetPayload_12()), uintAsFloat(optixGetPayload_13()), uintAsFloat(optixGetPayload_14()));
	payload.done = uintAsFloat(optixGetPayload_15());
	payload.seed = optixGetPayload_16();

	return payload;
}

static __forceinline__ __device__ void trace(RayPayload& payload) {
	unsigned int p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, p16;
	p0 = floatAsUint(payload.color.x);
	p1 = floatAsUint(payload.color.y);
	p2 = floatAsUint(payload.color.z);
	p3 = floatAsUint(payload.emittedLight.x);
	p4 = floatAsUint(payload.emittedLight.y);
	p5 = floatAsUint(payload.emittedLight.z);
	p6 = floatAsUint(payload.origin.x);
	p7 = floatAsUint(payload.origin.y);
	p8 = floatAsUint(payload.origin.z);
	p9 = floatAsUint(payload.direction.x);
	p10 = floatAsUint(payload.direction.y);
	p11 = floatAsUint(payload.direction.z);
	p12 = floatAsUint(payload.normal.x);
	p13 = floatAsUint(payload.normal.y);
	p14 = floatAsUint(payload.normal.z);
	p15 = floatAsUint(payload.done);
	p16 = payload.seed;

	optixTrace(params.handle, payload.origin, payload.direction, 0.0001f, 1e16f, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES, 0, 1, 0, p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14, p15, p16);

	payload.color = make_float3(uintAsFloat(p0), uintAsFloat(p1), uintAsFloat(p2));
	payload.emittedLight = make_float3(uintAsFloat(p3), uintAsFloat(p4), uintAsFloat(p5));
	payload.origin = make_float3(uintAsFloat(p6), uintAsFloat(p7), uintAsFloat(p8));
	payload.direction = make_float3(uintAsFloat(p9), uintAsFloat(p10), uintAsFloat(p11));
	payload.normal = make_float3(uintAsFloat(p12), uintAsFloat(p13), uintAsFloat(p14));
	payload.done = uintAsFloat(p15);
	payload.seed = p16;
}

extern "C" __global__ void __raygen__rg() {
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();
	unsigned int pixelIndex = idx.y * params.width + idx.x;
	float2 pixelOffset = make_float2(0.0f, 0.0f);
	RayPayload payload = {};
	payload.seed = pixelIndex;
	float3 totalLight = make_float3(0.0f, 0.0f, 0.0f);

	for (int i = 0; i < params.samplesPerPixel; i++) {
		payload.color = make_float3(1.0f);
		payload.emittedLight = make_float3(0.0f);
		payload.done = 0.0f;
		computeRay(idx, dim, payload.origin, payload.direction, pixelOffset);
		pixelOffset = make_float2(-0.5f + randomFloat(&payload.seed), -0.5f + randomFloat(&payload.seed));

		for (int j = 0; j < params.maxDepth; j++) {
			trace(payload);

			if (payload.done == 1.0f) {
				break;
			}
		}

		totalLight += payload.emittedLight;
	}

	params.image[pixelIndex] = totalLight / params.samplesPerPixel;
}

extern "C" __global__ void __closesthit__ch() {
	RayPayload payload = loadPayload();
	HitgroupData* hitData = (HitgroupData*)optixGetSbtDataPointer();
	/*
	float3 hitVertices[3];
	optixGetTriangleVertexData(optixGetGASTraversableHandle(), optixGetPrimitiveIndex(), optixGetSbtGASIndex(), optixGetRayTime(), hitVertices);
	payload.normal = normalize(cross(hitVertices[1] - hitVertices[0], hitVertices[2] - hitVertices[0]));
	*/
	unsigned int primitiveIndex = optixGetPrimitiveIndex();
	uint3 normalIndexTriplet = hitData->vertexNormalIndices[primitiveIndex];

	float3 normal0 = hitData->vertexNormals[normalIndexTriplet.x];
	float3 normal1 = hitData->vertexNormals[normalIndexTriplet.y];
	float3 normal2 = hitData->vertexNormals[normalIndexTriplet.z];

	float2 barycentrics = optixGetTriangleBarycentrics();
	float alpha = 1.0f - barycentrics.x - barycentrics.y;

	payload.normal = normal0 * alpha + normal1 * barycentrics.x + normal2 * barycentrics.y;

	payload.origin = payload.origin + optixGetRayTmax() * payload.direction;

	float3 diffuseDirection = normalize(payload.normal + randomDirection(&payload.seed));
	float3 specularDirection = payload.direction - 2 * dot(payload.direction, payload.normal) * payload.normal;
	bool isSpecualarBounce = hitData->metallic >= randomFloat(&payload.seed);
	payload.direction = normalize(lerp(specularDirection, diffuseDirection, hitData->roughness * (1 - isSpecualarBounce)));

	payload.emittedLight += (hitData->emissionColor * hitData->emissionPower) * payload.color;
	float3 color = lerp(hitData->color, make_float3(1.0f, 1.0f, 1.0f), isSpecualarBounce);
	payload.color *= color;

	payload.done = hitData->emissionPower > 0.0f ? 1.0f : 0.0f;

	storePayload(payload);
}

extern "C" __global__ void __miss__ms() {
	RayPayload payload = loadPayload();
	MissData* missData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
	
	float skyGradientT = pow(smoothstep(0.0f, 0.4f, payload.direction.y), 0.35f);
	float3 skyGradient = lerp(missData->skyColorHorizon, missData->skyColorZenith, skyGradientT);

	float groundToSkyT = smoothstep(-0.01f, 0.0f, payload.direction.y);

	float sun = pow(maximum(0, dot(payload.direction, -missData->sunDirection)), missData->sunFocus) * missData->sunIntensity;
	float sunMask = groundToSkyT >= 1;

	float3 light = lerp(missData->groundColor, skyGradient, groundToSkyT) + sun * sunMask;

	payload.emittedLight += make_float3(0.0f, 0.0f, 0.0f);
	payload.normal = make_float3(0.0f, 0.0f, 0.0f);
	payload.done = 1.0f;

	storePayload(payload);
}
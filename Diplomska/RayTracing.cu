#include "hip/hip_runtime.h"
#include <optix/optix.h>
#include "Parameters.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

struct RayPayload {
	float3 color;
	float3 emittedLight;
	float3 origin;
	float3 direction;
	float done;
	float prepForDenoiser;
	unsigned int seed;
};

extern "C" {
	__constant__ Params params;
}

static __forceinline__ __device__ void computeRay(uint3 idx, uint3 dim, float3& origin, float3& direction, float2& offset) {
	origin = params.camPosition;

	const mat4 inverseProjection = params.inverseProjection;
	const mat4 inverseView = params.inverseView;
	const float2 d = 2.0f * make_float2(
		(static_cast<float>(idx.x) + offset.x) / static_cast<float>(dim.x),
		(static_cast<float>(idx.y) + offset.y) / static_cast<float>(dim.y)
	) - 1.0f;

	float4 homogeniousDeviceCoords = make_float4(d.x, -d.y, 1.0f, 1.0f);
	float4 viewSpace = inverseProjection * homogeniousDeviceCoords;
	float4 worldSpace = inverseView * make_float4(viewSpace.x, viewSpace.y, viewSpace.z, 0.0f);

	direction = make_float3(normalize(worldSpace));
}

static __forceinline__ __device__ void storePayload(RayPayload payload) {
	optixSetPayload_0(floatAsUint(payload.color.x));
	optixSetPayload_1(floatAsUint(payload.color.y));
	optixSetPayload_2(floatAsUint(payload.color.z));

	optixSetPayload_3(floatAsUint(payload.emittedLight.x));
	optixSetPayload_4(floatAsUint(payload.emittedLight.y));
	optixSetPayload_5(floatAsUint(payload.emittedLight.z));

	optixSetPayload_6(floatAsUint(payload.origin.x));
	optixSetPayload_7(floatAsUint(payload.origin.y));
	optixSetPayload_8(floatAsUint(payload.origin.z));

	optixSetPayload_9(floatAsUint(payload.direction.x));
	optixSetPayload_10(floatAsUint(payload.direction.y));
	optixSetPayload_11(floatAsUint(payload.direction.z));

	optixSetPayload_12(floatAsUint(payload.done));
	optixSetPayload_13(payload.seed);
	optixSetPayload_14(payload.prepForDenoiser);
}

static __forceinline__ __device__ RayPayload loadPayload() {
	RayPayload payload = {};

	payload.color = make_float3(uintAsFloat(optixGetPayload_0()), uintAsFloat(optixGetPayload_1()), uintAsFloat(optixGetPayload_2()));
	payload.emittedLight = make_float3(uintAsFloat(optixGetPayload_3()), uintAsFloat(optixGetPayload_4()), uintAsFloat(optixGetPayload_5()));
	payload.origin = make_float3(uintAsFloat(optixGetPayload_6()), uintAsFloat(optixGetPayload_7()), uintAsFloat(optixGetPayload_8()));
	payload.direction = make_float3(uintAsFloat(optixGetPayload_9()), uintAsFloat(optixGetPayload_10()), uintAsFloat(optixGetPayload_11()));
	payload.done = uintAsFloat(optixGetPayload_12());
	payload.seed = optixGetPayload_13();
	payload.prepForDenoiser = uintAsFloat(optixGetPayload_14());

	return payload;
}

static __forceinline__ __device__ void trace(RayPayload& payload) {
	unsigned int p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14;
	p0 = floatAsUint(payload.color.x);
	p1 = floatAsUint(payload.color.y);
	p2 = floatAsUint(payload.color.z);
	p3 = floatAsUint(payload.emittedLight.x);
	p4 = floatAsUint(payload.emittedLight.y);
	p5 = floatAsUint(payload.emittedLight.z);
	p6 = floatAsUint(payload.origin.x);
	p7 = floatAsUint(payload.origin.y);
	p8 = floatAsUint(payload.origin.z);
	p9 = floatAsUint(payload.direction.x);
	p10 = floatAsUint(payload.direction.y);
	p11 = floatAsUint(payload.direction.z);
	p12 = floatAsUint(payload.done);
	p13 = payload.seed;
	p14 = floatAsUint(payload.prepForDenoiser);

	optixTrace(params.handle, payload.origin, payload.direction, 0.0001f, 1e16f, 0.0f, OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 1, 0, p0, p1, p2, p3, p4, p5, p6, p7, p8, p9, p10, p11, p12, p13, p14);

	payload.color = make_float3(uintAsFloat(p0), uintAsFloat(p1), uintAsFloat(p2));
	payload.emittedLight = make_float3(uintAsFloat(p3), uintAsFloat(p4), uintAsFloat(p5));
	payload.origin = make_float3(uintAsFloat(p6), uintAsFloat(p7), uintAsFloat(p8));
	payload.direction = make_float3(uintAsFloat(p9), uintAsFloat(p10), uintAsFloat(p11));
	payload.done = uintAsFloat(p12);
	payload.seed = p13;
	payload.prepForDenoiser = p14;
}

extern "C" __global__ void __raygen__rg() {
	const uint3 idx = optixGetLaunchIndex();
	const uint3 dim = optixGetLaunchDimensions();
	unsigned int pixelIndex = idx.y * params.width + idx.x;
	float2 pixelOffset = make_float2(0.0f, 0.0f);
	RayPayload payload = {};
	payload.seed = pixelIndex + params.frameIndex * 719393;
	payload.prepForDenoiser = 1.0f;

	float3 totalLight = make_float3(0.0f, 0.0f, 0.0f);

	for (int i = 0; i < params.samplesPerPixel; i++) {
		payload.color = make_float3(1.0f);
		payload.emittedLight = make_float3(0.0f);
		payload.done = 0.0f;
		computeRay(idx, dim, payload.origin, payload.direction, pixelOffset);
		pixelOffset = make_float2(-0.5f + randomFloat(&payload.seed), -0.5f + randomFloat(&payload.seed));

		for (int j = 0; j < params.maxDepth; j++) {
			trace(payload);
			payload.prepForDenoiser = 0.0f;

			if (payload.done == 1.0f) {
				break;
			}
		}

		totalLight += payload.emittedLight;
	}

	float3 averageLight = totalLight / params.samplesPerPixel;
	float weight = 1.0f / (params.frameIndex + 1);

	params.image[pixelIndex] = params.image[pixelIndex] * (1.0f - weight) + averageLight * weight;
}

extern "C" __global__ void __closesthit__ch() {
	RayPayload payload = loadPayload();
	HitgroupData* hitData = (HitgroupData*)optixGetSbtDataPointer();
	/*
	float3 hitVertices[3];
	optixGetTriangleVertexData(optixGetGASTraversableHandle(), optixGetPrimitiveIndex(), optixGetSbtGASIndex(), optixGetRayTime(), hitVertices);
	float3 normal = normalize(cross(hitVertices[1] - hitVertices[0], hitVertices[2] - hitVertices[0]));
	*/
	unsigned int primitiveIndex = optixGetPrimitiveIndex();
	uint3 normalIndexTriplet = hitData->vertexNormalIndices[primitiveIndex];

	float3 normal0 = hitData->vertexNormals[normalIndexTriplet.x];
	float3 normal1 = hitData->vertexNormals[normalIndexTriplet.y];
	float3 normal2 = hitData->vertexNormals[normalIndexTriplet.z];

	float2 barycentrics = optixGetTriangleBarycentrics();
	float alpha = 1.0f - barycentrics.x - barycentrics.y;

	float3 normal = normal0 * alpha + normal1 * barycentrics.x + normal2 * barycentrics.y;
	
	payload.origin = payload.origin + optixGetRayTmax() * payload.direction;

	float3 diffuseDirection = normalize(normal + randomDirection(&payload.seed));
	float3 specularDirection = payload.direction - 2 * dot(payload.direction, normal) * normal;
	bool isSpecualarBounce = hitData->metallic >= randomFloat(&payload.seed);
	payload.direction = normalize(lerp(specularDirection, diffuseDirection, hitData->roughness * (1 - isSpecualarBounce)));

	payload.emittedLight += (hitData->emissionColor * hitData->emissionPower) * payload.color;
	float3 color = lerp(hitData->color, make_float3(1.0f, 1.0f, 1.0f), isSpecualarBounce);
	payload.color *= color;

	payload.done = hitData->emissionPower > 0.0f ? 1.0f : 0.0f;

	if (payload.prepForDenoiser) {
		const uint3 idx = optixGetLaunchIndex();
		const uint3 dim = optixGetLaunchDimensions();
		unsigned int pixelIndex = idx.y * params.width + idx.x;

		params.normals[pixelIndex] = normal;
		params.albedo[pixelIndex] = hitData->color;
	}

	storePayload(payload);
}

extern "C" __global__ void __miss__ms() {
	RayPayload payload = loadPayload();
	MissData* missData = reinterpret_cast<MissData*>(optixGetSbtDataPointer());
	/*
	float skyGradientT = pow(smoothstep(0.0f, 0.4f, payload.direction.y), 0.35f);
	float3 skyGradient = lerp(missData->skyColorHorizon, missData->skyColorZenith, skyGradientT);

	float groundToSkyT = smoothstep(-0.01f, 0.0f, payload.direction.y);

	float sun = pow(maximum(0, dot(payload.direction, -missData->sunDirection)), missData->sunFocus) * missData->sunIntensity;
	float sunMask = groundToSkyT >= 1;

	float3 light = lerp(missData->groundColor, skyGradient, groundToSkyT) + sun * sunMask;
	*/
	payload.emittedLight += make_float3(0.0f, 0.0f, 0.0f);
	payload.done = 1.0f;

	if (payload.prepForDenoiser) {
		const uint3 idx = optixGetLaunchIndex();
		const uint3 dim = optixGetLaunchDimensions();
		unsigned int pixelIndex = idx.y * params.width + idx.x;

		params.normals[pixelIndex] = make_float3(0.0f);
		params.albedo[pixelIndex] = payload.emittedLight;
	}

	storePayload(payload);
}